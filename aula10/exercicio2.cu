#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>   // Biblioteca para vetores na GPU
#include <thrust/host_vector.h>     // Biblioteca para vetores na CPU
#include <thrust/reduce.h>          // Biblioteca para operações de redução
#include <thrust/extrema.h>         // Biblioteca para encontrar o máximo e o mínimo
#include <iostream>                 // Biblioteca padrão para entrada e saída de dados
#include <fstream>                  // Biblioteca para manipulação de arquivos
#include <chrono>                   // Biblioteca para medir o tempo de execução

int main() {
    std::ifstream file("stocks-google.txt");  // Abre o arquivo
    if (!file) {
        std::cerr << "Erro ao abrir o arquivo!" << std::endl;
        return 1;
    }

    // Contagem do número de linhas no arquivo
    int n = 0;
    std::string line;
    while (std::getline(file, line)) {
        n++;
    }

    // Reinicia o ponteiro de leitura para o início do arquivo
    file.clear();
    file.seekg(0, std::ios::beg);

    // Variável para armazenar o valor lido
    double value = 0.0;

    // Criação do host_vector para armazenar os dados na CPU
    thrust::host_vector<double> host(n, 0);

    // Leitura dos valores do arquivo e armazenamento no host_vector
    for (int i = 0; i < n; i++) {
        file >> value;  // Lê o valor do arquivo
        host[i] = value;    // Armazena no host_vector
    }

    // Fecha o arquivo
    file.close();

    // Transferir dados para a GPU
    thrust::device_vector<double> device(host);

    // Cálculo do preço médio nos últimos 10 anos (período total)
    double soma_total = thrust::reduce(device.begin(), device.end(), 0.0, thrust::plus<double>());
    double media_total = soma_total / n;
    std::cout << "Preço médio das ações nos últimos 10 anos: " << media_total << std::endl;

    // Cálculo do preço médio nos últimos 365 dias
    int ultimo_ano = std::min(365, n);  // Caso o número de dias seja menor que 365
    double soma_ultimo_ano = thrust::reduce(device.end() - ultimo_ano, device.end(), 0.0, thrust::plus<double>());
    double media_ultimo_ano = soma_ultimo_ano / ultimo_ano;
    std::cout << "Preço médio das ações nos últimos 365 dias: " << media_ultimo_ano << std::endl;

    // Encontrar o maior e menor preço no período total
    auto minmax_total = thrust::minmax_element(device.begin(), device.end());
    double menor_total = *minmax_total.first;
    double maior_total = *minmax_total.second;
    std::cout << "Menor preço nos últimos 10 anos: " << menor_total << std::endl;
    std::cout << "Maior preço nos últimos 10 anos: " << maior_total << std::endl;

    // Encontrar o maior e menor preço nos últimos 365 dias
    auto minmax_ultimo_ano = thrust::minmax_element(device.end() - ultimo_ano, device.end());
    double menor_ultimo_ano = *minmax_ultimo_ano.first;
    double maior_ultimo_ano = *minmax_ultimo_ano.second;
    std::cout << "Menor preço nos últimos 365 dias: " << menor_ultimo_ano << std::endl;
    std::cout << "Maior preço nos últimos 365 dias: " << maior_ultimo_ano << std::endl;

    return 0;
}
